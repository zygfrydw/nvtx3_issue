
#include <hip/hip_runtime.h>
#include <thread>
#include <chrono>
#include <iostream>
#include "nvtx3/nvtx3.hpp"

using namespace std::chrono_literals;

void __global__ kernel(float* a, size_t n){
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n){
        a[tid] = a[tid] + 10.0;
    }
}

struct my_domain{ static constexpr char const* name{"my_domain"}; };

void function_my_domain(){
    nvtx3::scoped_range_in<my_domain> r(__FUNCTION__);
    std::cout << "function_my_domain\n";
    size_t N = 10'000'000; 
    float * a;
    hipMalloc(&a, N * sizeof(float));    
    kernel<<<N, 1>>>(a, N);
    hipFree(a);
    std::this_thread::sleep_for(1s);
}

void function_global(){
    nvtx3::scoped_range r(__FUNCTION__);
    std::cout << "function_global\n";
    size_t N = 10'000'000; 
    float * a;
    hipMalloc(&a, N * sizeof(float));    
    kernel<<<N, 1>>>(a, N);
    hipFree(a);
    std::this_thread::sleep_for(1s);
}


int main(){
    function_my_domain();
    function_global();
    return 0;
}